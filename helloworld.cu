
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void hello_world_gpu()
{
    printf("Hello World from the GPU\n");
}

int main(void)
{
    hello_world_gpu<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}